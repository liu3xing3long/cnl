#include "hip/hip_runtime.h"
#include "..\Global\stdafx.h"

__global__ void executeLayerKernel(const real_gpu *dp_pLayerInput,const real_gpu *dp_pWeights,real_gpu *dp_pLayerOutput,real_gpu *dp_pDerivativeOfLastOutput,int p_iNumInputNeurons, Neuron::NeuronType p_eNeuronType,int p_iOutputNeuronCount)
{
	extern __shared__ real_gpu s_InputNeurons[];
	real_gpu* s_InputWeights = &s_InputNeurons[p_iNumInputNeurons];

	int iNumInputNeuronsAligned = ALIGN_UP(p_iNumInputNeurons, HALF_WARP);
	int iNumOutputNeuronsAligned = ALIGN_UP(blockDim.x, HALF_WARP);
	
	const real_gpu *d_LayerInputThisTest = dp_pLayerInput + blockIdx.x*iNumInputNeuronsAligned;
	int iMoveWeightsForThisTest = threadIdx.x*p_iNumInputNeurons;
	const real_gpu *d_WeightsThisTest = dp_pWeights + iMoveWeightsForThisTest;
	real_gpu *d_pLayerOutputThisTest = dp_pLayerOutput + blockIdx.x*iNumOutputNeuronsAligned + threadIdx.x;
	real_gpu *d_pDerivativeOfLastOutputThisTest = dp_pDerivativeOfLastOutput + blockIdx.x*iNumOutputNeuronsAligned + threadIdx.x;

	// first, we copy d_LayerInputThisTest to s_InputNeurons
	for(int iInputIndex = threadIdx.x;iInputIndex < p_iNumInputNeurons; iInputIndex+=blockDim.x)
	{
		s_InputNeurons[iInputIndex] = d_LayerInputThisTest[iInputIndex];
	}

	// we have to make sure that all data was written to shared memory
	__syncthreads();

	real_gpu dResult = 0.0f;
	
	/*if(threadIdx.x == 1 && blockIdx.x == 1)
	{
		printf("INPUT %d | WEIGHTS %d | OUTPUT %d\n",d_LayerInputThisTest - dp_pLayerInput,d_WeightsThisTest - dp_pWeights,d_pLayerOutputThisTest - dp_pLayerOutput);
	}*/	

	int iNumOfWeights = p_iNumInputNeurons * p_iOutputNeuronCount;
	int iNumOfWeightsAligned = ALIGN_UP(iNumOfWeights,blockDim.x);
	for(int iWeightIndex = threadIdx.x, iWeightIndexBase = 0 ; iWeightIndex < iNumOfWeightsAligned ; iWeightIndex += blockDim.x, iWeightIndexBase += blockDim.x)
	{
		// first, we copy d_WeightsThisTest to s_InputWeights (it is only a part of weights)
		if(iWeightIndex < iNumOfWeights)
		{
			s_InputWeights[threadIdx.x] = d_WeightsThisTest[iWeightIndex];
		}

		int iFirstElementInThisBatch = iMoveWeightsForThisTest - iWeightIndexBase;
		int iLastElementInThisBatch = iFirstElementInThisBatch + p_iNumInputNeurons;

		// Not all threads are used in calulations
		if(threadIdx.x < p_iOutputNeuronCount && iLastElementInThisBatch < iWeightIndexBase+p_iOutputNeuronCount && iMoveWeightsForThisTest+iNumInputNeuronsAligned >= iWeightIndexBase)
		{
			int iFirstWeightIndex = min(0,iWeightIndexBase - iMoveWeightsForThisTest);
			int iLastWeightIndex = max(p_iNumInputNeurons,iWeightIndexBase + p_iOutputNeuronCount - iMoveWeightsForThisTest);
			for(int iWeightIndex = iFirstWeightIndex;iWeightIndex < iLastWeightIndex; ++iWeightIndex)
			{
				int iWeightIndexHere = iWeightIndex + iWeightIndexBase - iMoveWeightsForThisTest;
				PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d , iWeightIndex %d : d_LayerInputThisTest %f , d_WeightsThisTest %f , iWeightIndexHere %d, val[%d] %f , MULT %f\n",blockIdx.x,threadIdx.x,iWeightIndex,d_LayerInputThisTest[iWeightIndex],d_WeightsThisTest[iWeightIndex],iWeightIndexHere,iWeightIndexHere,s_InputWeights[iWeightIndexHere],d_LayerInputThisTest[iWeightIndex] * d_WeightsThisTest[iWeightIndex]);
				dResult += s_InputNeurons[iWeightIndex] * s_InputWeights[iWeightIndexHere];
			}
		}
	}

	if(threadIdx.x <= p_iOutputNeuronCount)
	{
		double dDerivativeOfLastOutput = 0.0f;

		PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d : dResult before output function %f\n",blockIdx.x,threadIdx.x,dResult);

		switch(p_eNeuronType)
		{		
			case Neuron::NT_LINEAR: 
				dDerivativeOfLastOutput = 1.0f;
				break;	// Do nothing
			case Neuron::NT_SIGMOID: 
				double dExp = exp(-dResult);
				dResult = 1.0 / (1.0 + dExp);
				dDerivativeOfLastOutput = dExp / pow(1.0 + dExp,2);
				break;	
		}
		
		if(threadIdx.x == p_iOutputNeuronCount)
			dResult = 1.0f; /* bias */
			
		*d_pLayerOutputThisTest = dResult;
		
		// We only need derivative of last output if we are in training!
		if(dp_pDerivativeOfLastOutput != NULL)
			*d_pDerivativeOfLastOutputThisTest = dDerivativeOfLastOutput;

		PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d : first d_LayerInputThisTest %f , first d_WeightsThisTest %f , dResult %f , dDerivativeOfLastOutput %f\n",blockIdx.x,threadIdx.x,d_LayerInputThisTest[0],d_WeightsThisTest[0],dResult,dDerivativeOfLastOutput);
	}
}

extern "C" void executeLayerCUDA(const real_gpu *dp_pLayerInput,const real_gpu *dp_pWeights,real_gpu *dp_pLayerOutput,real_gpu *dp_pDerivativeOfLastOutput,int p_iTestCount,int p_iOutputNeuronCount,int p_iNumInputNeurons,Neuron::NeuronType p_eNeuronType)
{
	// blockDim.x should be a multiple of 16 (half warp). We will be able to retrieve global data using coalescing
	int iBlockDimUpdated = ALIGN_UP(p_iOutputNeuronCount+1,HALF_WARP);
	int iSharedMemorySize = p_iNumInputNeurons * sizeof(real_gpu); // memory for input
	iSharedMemorySize += iBlockDimUpdated * sizeof(real_gpu); // memory for weights
	executeLayerKernel <<<p_iTestCount,iBlockDimUpdated,iSharedMemorySize>>> (dp_pLayerInput,dp_pWeights,dp_pLayerOutput,dp_pDerivativeOfLastOutput,p_iNumInputNeurons,p_eNeuronType,p_iOutputNeuronCount);
}
