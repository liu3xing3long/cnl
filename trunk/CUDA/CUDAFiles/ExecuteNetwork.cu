#include "hip/hip_runtime.h"
#include "..\Global\stdafx.h"

__global__ void executeLayerKernel(const real_gpu *dp_pLayerInput,const real_gpu *dp_pWeights,real_gpu *dp_pLayerOutput,real_gpu *dp_pDerivativeOfLastOutput,int p_iNumInputNeurons, Neuron::NeuronType p_eNeuronType,int p_iOutputNeuronCount)
{
	extern __shared__ real_gpu sharedInputNeurons[];

	int iNumInputNeuronsAligned = ALIGN_UP(p_iNumInputNeurons, HALF_WARP);
	int iNumOutputNeuronsAligned = ALIGN_UP(blockDim.x, HALF_WARP);
	
	const real_gpu *d_LayerInputThisTest = dp_pLayerInput + blockIdx.x*iNumInputNeuronsAligned;
	const real_gpu *d_WeightsThisTest = dp_pWeights + threadIdx.x*p_iNumInputNeurons;
	real_gpu *d_pLayerOutputThisTest = dp_pLayerOutput + blockIdx.x*iNumOutputNeuronsAligned + threadIdx.x;
	real_gpu *d_pDerivativeOfLastOutputThisTest = dp_pDerivativeOfLastOutput + blockIdx.x*iNumOutputNeuronsAligned + threadIdx.x;

	// first, we copy d_LayerInputThisTest to sharedInputNeurons
	/*for(int iInputIndex = threadIdx.x;iInputIndex < p_iNumInputNeurons; iInputIndex+=blockDim.x)
	{
		sharedInputNeurons[iInputIndex] = d_LayerInputThisTest[iInputIndex];
	}*/

	// we have to make sure that all data was written to shared memory
	__syncthreads();

	// Not all threads are used in calulations
	if(threadIdx.x <= p_iOutputNeuronCount)
	{	
		real_gpu dResult = 0.0f;
		
		/*if(threadIdx.x == 1 && blockIdx.x == 1)
		{
			printf("INPUT %d | WEIGHTS %d | OUTPUT %d\n",d_LayerInputThisTest - dp_pLayerInput,d_WeightsThisTest - dp_pWeights,d_pLayerOutputThisTest - dp_pLayerOutput);
		}*/	
		
		for(int iWeightIndex = 0;iWeightIndex < p_iNumInputNeurons; ++iWeightIndex)
		{
			PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d , iWeightIndex %d : d_LayerInputThisTest %f , d_WeightsThisTest %f , MULT %f\n",blockIdx.x,threadIdx.x,iWeightIndex,d_LayerInputThisTest[iWeightIndex],d_WeightsThisTest[iWeightIndex],d_LayerInputThisTest[iWeightIndex] * d_WeightsThisTest[iWeightIndex]);
			dResult += d_LayerInputThisTest[iWeightIndex] * d_WeightsThisTest[iWeightIndex];
		}
		
		double dDerivativeOfLastOutput = 0.0f;

		PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d : dResult before output function %f\n",blockIdx.x,threadIdx.x,dResult);

		switch(p_eNeuronType)
		{		
			case Neuron::NT_LINEAR: 
				dDerivativeOfLastOutput = 1.0f;
				break;	// Do nothing
			case Neuron::NT_SIGMOID: 
				double dExp = exp(-dResult);
				dResult = 1.0 / (1.0 + dExp);
				dDerivativeOfLastOutput = dExp / pow(1.0 + dExp,2);
				break;	
		}
		
		if(threadIdx.x == p_iOutputNeuronCount)
			dResult = 1.0f; /* bias */
			
		*d_pLayerOutputThisTest = dResult;
		
		// We only need derivative of last output if we are in training!
		if(dp_pDerivativeOfLastOutput != NULL)
			*d_pDerivativeOfLastOutputThisTest = dDerivativeOfLastOutput;

		PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d : first d_LayerInputThisTest %f , first d_WeightsThisTest %f , dResult %f , dDerivativeOfLastOutput %f\n",blockIdx.x,threadIdx.x,d_LayerInputThisTest[0],d_WeightsThisTest[0],dResult,dDerivativeOfLastOutput);
	}
}

extern "C" void executeLayerCUDA(const real_gpu *dp_pLayerInput,const real_gpu *dp_pWeights,real_gpu *dp_pLayerOutput,real_gpu *dp_pDerivativeOfLastOutput,int p_iTestCount,int p_iOutputNeuronCount,int p_iNumInputNeurons,Neuron::NeuronType p_eNeuronType)
{
	// blockDim.x should be a multiple of 16 (half warp). We will be able to retrieve global data using coalescing
	int iBlockDimUpdated = ALIGN_UP(p_iOutputNeuronCount+1,HALF_WARP);
	int iSharedMemorySize = p_iNumInputNeurons * sizeof(real_gpu);
	executeLayerKernel <<<p_iTestCount,iBlockDimUpdated,iSharedMemorySize>>> (dp_pLayerInput,dp_pWeights,dp_pLayerOutput,dp_pDerivativeOfLastOutput,p_iNumInputNeurons,p_eNeuronType,p_iOutputNeuronCount);
}
